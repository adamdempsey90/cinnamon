#include "hip/hip_runtime.h"
/* 1D viscous diffusion problem
 *
 * The self-similar solution of the density at any time is
 *
 * d(x,t) = d0/sqrt(pi s) exp(-x^2/s)
 * where s = 4 * nu * (t-t0)
 *
 *  */
#include "defs.h"
#include <time.h>
#include "cuda_defs.h"

__device__ __managed__ real nu = .01;


__device__ real kinematic_viscosity(real x1, real x2, real x3) {
    return nu;
}

/* Select boundary conditions */
__device__ void x1_boundary_inner(int indxg, int i, int j,int k, real *cons, real *intenergy, real *x1, real *x2, real *x3, int nx1, int nx2, int nx3, int ntot, int nf, int size_x1, int size_x12, int offset, real g, real time) {
	//outflow_boundary_inner(1,indxg,i,j,k,cons,intenergy,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
	return;
}
__device__ void x1_boundary_outer(int indxg, int i, int j,int k, real *cons, real *intenergy, real *x1, real *x2, real *x3, int nx1, int nx2, int nx3, int ntot, int nf, int size_x1, int size_x12, int offset, real g, real time) {
	//outflow_boundary_outer(1,indxg,i,j,k,cons,intenergy,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
	return;
}

//extern "C" {
void scale_factors(real x1, real x2, real x3, real *h1, real *h2, real *h3) {
    *h1 = 1.;
    *h2 = 1.;
    *h3 = 1.;
}
void init_mesh(GridCons *grid, Parameters *params) {
    
    init_uniform_mesh(grid,params);

    return;

}
void init_gas(GridCons *grid, Parameters *params) {
    int i,j,k,indx;
    int nx1,nx2,nx3,n,ntot,nf;
    int size_x1,size_x12; 
    nx1 = grid->nx[0];
    nx2 = grid->nx[1];
    nx3 = grid->nx[2];
    size_x1 = grid->size_x1;
    size_x12 = grid->size_x12;
    ntot = grid->ntot;
    nf = grid->nf;

    nu = params->nu;
    real w = params->width;
    real dinit = params->dinit;



    real *x1 = grid->xc1;

    real *rho       = &grid->cons[0*ntot];
    real *mx1       = &grid->cons[1*ntot];
    real *mx2       = &grid->cons[2*ntot];
    real *mx3       = &grid->cons[3*ntot];
    real *energy    = &grid->cons[4*ntot];
    real *intenergy = grid->intenergy; 

    real gamma = params->gamma ;

    real ke;

    real u1 = 0;
    real u2 = 0;
    real u3 = 0;
    real pres;
    real t = .1;



    for(k=-NGHX3;k<nx3+NGHX3;k++) {
		for(j=-NGHX2;j<nx2+NGHX2;j++) {
			for(i=-NGHX1;i<nx1+NGHX1;i++) {
				indx = INDEX(i,j,k);


				// Set gaussian temp ic.
				rho[indx] = (dinit-1)*exp(-x1[i]*x1[i]/(4*nu*t))/sqrt(4*M_PI*nu*t) + 1.;

				pres = 1.;

				mx1[indx] = u1*rho[indx];
				mx2[indx] = u2*rho[indx];
				mx3[indx] = u3*rho[indx];

				ke = mx1[indx]*mx1[indx] + mx2[indx]*mx2[indx] + mx3[indx]*mx3[indx];
				ke /= 2*rho[indx];
				intenergy[indx] = pres/ (gamma-1);
				energy[indx] = intenergy[indx] + ke;
				for(n=5;n<nf;n++) {
					grid->cons[n*ntot+indx] = 0;
				}





			}
		}
    }
    return;


}
//}
